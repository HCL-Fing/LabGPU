#include "hip/hip_runtime.h"
#include "util.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

using namespace std;

__global__ void blur_kernel(float* d_input, float* d_output, float* d_msk, int width, int height){

}

__global__ void ajustar_brillo_coalesced_kernel(float* d_input, float* d_output, int width, int height, float coef){

}

__global__ void ajustar_brillo_no_coalesced_kernel(float* d_input, float* d_output, int width, int height, float coef){

}

void ajustar_brillo_gpu(float * img_in, int width, int height, float * img_out, float coef, int filas=1){
    
    // Reservar memoria en la GPU

    // copiar imagen y máscara a la GPU
   
    // configurar grilla y lanzar kernel
   
    // transferir resultado a la memoria principal

    // liberar la memoria
}


void blur_gpu(float * img_in, int width, int height, float * img_out, float msk[], int m_size){
    
    // Reservar memoria en la GPU

    // copiar imagen y máscara a la GPU
   
    // configurar grilla y lanzar kernel
   
    // transferir resultado a la memoria principal

	// liberar la memoria
}

void ajustar_brillo_cpu(float * img_in, int width, int height, float * img_out, float coef){

    CLK_POSIX_INIT;
    CLK_POSIX_START;

    for(int imgx=0; imgx < width ; imgx++){
        for(int imgy=0; imgy < height; imgy++){
            img_out[imgy*width+imgx] = min(255.0f,max(0.0f,img_in[imgy*width+imgx]+coef));
        }
    }

    CLK_POSIX_STOP;
    CLK_POSIX_ELAPSED;

    printf("Tiempo ajustar brillo CPU: %f ms\n", t_elap);
}

void blur_cpu(float * img_in, int width, int height, float * img_out, float msk[], int m_size){

    CLK_POSIX_INIT;
    CLK_POSIX_START;

    float val_pixel=0;
    
    //para cada pixel aplicamos el filtro
    for(int imgx=0; imgx < width ; imgx++){
        for(int imgy=0; imgy < height; imgy++){

            val_pixel = 0;

            // aca aplicamos la mascara
            for (int i = 0; i < m_size ; i++){
                for (int j = 0; j < m_size ; j++){
                    
                    int ix =imgx + i - m_size/2;
                    int iy =imgy + j - m_size/2;
                    
                    if(ix >= 0 && ix < width && iy>= 0 && iy < height )
                        val_pixel = val_pixel +  img_in[iy * width +ix] * msk[i*m_size+j];
                }
            }
            
            // guardo valor resultado
            img_out[imgy*width+imgx]= val_pixel;
        }
    }

    CLK_POSIX_STOP;
    CLK_POSIX_ELAPSED;

    printf("Tiempo filtro Gaussiano CPU: %f ms\n", t_elap);
}